#include "hip/hip_runtime.h"
#include "motis/tripbased/gpu/gpu_tripbased.h"

#include <cstdio>

namespace motis::tripbased {

#define CUDA_ALLOC(target, size) \
  hipMalloc((void**) &(target), size);

#define CUDA_COPY(target, source, size, copy_type) \
  hipMemcpy(target, source, size, copy_type);

#define CUDA_ALLOC_COPY(target, source, size)   \
  CUDA_ALLOC(target, size)                      \
  CUDA_COPY(target, source, size, hipMemcpyHostToDevice)

__global__ void check_dominated(gpu_device_pointers pointers) {
  unsigned idx1 = threadIdx.x;
  unsigned idx2 = threadIdx.y;
  if (idx1 <= idx2 || idx1 >= *pointers.result_set_size_device_) {
    return;
  }
  gpu_tb_journey journey1 = pointers.result_set_device_[idx1];
  gpu_tb_journey journey2 = pointers.result_set_device_[idx2];
  if (journey1.transfers_ == journey2.transfers_) {
    if (journey1.arrival_time_ > journey2.arrival_time_) {
      pointers.is_dominated_device_[idx1] = 1;
    } else {
      pointers.is_dominated_device_[idx2] = 1;
    }
  } else if (journey1.transfers_ > journey2.transfers_) {
    if(journey1.arrival_time_ > journey2.arrival_time_) {
      pointers.is_dominated_device_[idx1] = 1;
    }
  } else { // journey2.transfers_ < journey2.transfers_
    if (journey1.arrival_time_ < journey2.arrival_time_) {
      pointers.is_dominated_device_[idx2] = 1;
    }
  }
}

__device__ void destination_reached(gpu_device_pointers pointers,
                                    gpu_queue_entry queue_entry,
                                    unsigned entry_num,
                                    gpu_dest_arrival dest_arrival,
                                    unsigned transfers) {
  // TODO(sarah): max_travel_time limit check?
  unsigned set_entry_index = atomicAdd(pointers.result_set_size_device_, 1U);
  uint16_t arrival_time
      = pointers.arrival_times_device_.data_[pointers.arrival_times_device_
                                                  .index_[queue_entry.trip_]
                                              + dest_arrival.stop_index_]
        + dest_arrival.fp_duration_;
  pointers.result_set_device_[set_entry_index]
      = gpu_tb_journey{*pointers.start_time_device_,
                       arrival_time,
                       transfers,
                       transfers + 1,
                       dest_arrival.fp_to_station_id_,
                       dest_arrival,
                       entry_num};
  //std::printf("1\n");
}

__device__ void enqueue(gpu_device_pointers const pointers,
                        uint32_t trip,
                        uint16_t stop_idx,
                        unsigned transfers,
                        std::size_t prev_trip_seg) {
  auto const old_first_reachable = pointers.first_reachable_stop_device_[trip];
  if (stop_idx < old_first_reachable) {
    unsigned queue_entry_index = atomicAdd(&pointers.queue_sizes_device_[transfers+1], 1U);
    pointers.queue_device_[pointers.queue_index_device_[transfers + 1] + queue_entry_index]
        = gpu_queue_entry{trip, stop_idx, old_first_reachable, prev_trip_seg};
    auto const line = pointers.trip_to_line_device_[trip];
    for (uint32_t t = trip;
         t < *pointers.trip_count_device_ && pointers.trip_to_line_device_[t] == line;
         ++t) {
      if (stop_idx < pointers.first_reachable_stop_device_[t]) {
        pointers.first_reachable_stop_device_[t] = stop_idx;
      }
    }
  }
}

__global__ void search(gpu_device_pointers const pointers,
                       unsigned const transfers,
                       unsigned const max_transfers) {
  unsigned trip_seg = blockIdx.x * 32 + threadIdx.x;
  unsigned long idx = trip_seg + pointers.queue_index_device_[transfers];
  if (trip_seg >= pointers.queue_sizes_device_[transfers]) {
    return;
  }
  auto& entry = pointers.queue_device_[idx];
  auto const line = pointers.trip_to_line_device_[entry.trip_];

  auto const dest_arrs_size = pointers.dest_arrivals_index_device_[line + 1]
                                  - pointers.dest_arrivals_index_device_[line];
  if (dest_arrs_size > 0) {
    for (auto i = pointers.dest_arrivals_index_device_[line];
         i < pointers.dest_arrivals_index_device_[line + 1]; ++i) {
      if (entry.from_stop_index_ < pointers.dest_arrivals_device_[i].stop_index_) {
        destination_reached(pointers,
                            entry,
                            trip_seg,
                            pointers.dest_arrivals_device_[i],
                            transfers);
      }
    }
  }

  if (transfers + 1 > max_transfers) {
    return;
  }
  auto next_stop_arrival_times = pointers.arrival_times_device_
                                     .data_[pointers.arrival_times_device_
                                                .index_[entry.trip_]
                                            + entry.from_stop_index_ + 1];

  if (next_stop_arrival_times < *pointers.total_earliest_arrival_device_) {
    auto const stop_count =
        std::min(entry.to_stop_index_,
                 static_cast<stop_idx_t>(pointers.line_stop_count_device_[line] - 1));
    for (auto i = entry.from_stop_index_ + 1; i <= stop_count; ++i) {
      auto start_idx = pointers.transfers_device
                           .index_[pointers.transfers_device
                                       .base_index_[entry.trip_] + i];
      auto end_idx = pointers.transfers_device
                         .index_[pointers.transfers_device
                                     .base_index_[entry.trip_] + i + 1];
      for (auto j = start_idx; j < end_idx; ++j) {
        enqueue(pointers,
                pointers.transfers_device.data_[j].to_trip,
                pointers.transfers_device.data_[j].to_stop_idx,
                transfers,
                trip_seg);
      }
    }
  }
}

gpu_search_results search_fwd_gpu(unsigned const max_transfers,
                    gpu_device_pointers const pointers) {
  std::vector<unsigned> queue_sizes(max_transfers + 1);
  // TODO(sarah): <= or < ?
  for (auto transfers = 0U; transfers <= max_transfers; ++transfers) {
    CUDA_COPY(&queue_sizes[transfers],
              &pointers.queue_sizes_device_[transfers],
              sizeof(std::size_t),
              hipMemcpyDeviceToHost)
    // TODO(sarah)
    unsigned thread_num = queue_sizes[transfers];
    unsigned block_num = 1;
    if(thread_num > 32) {
      block_num = (thread_num + 31) / 32;
      thread_num = 32;
    }
    search<<<block_num, thread_num>>>
        (pointers, transfers, max_transfers);
    hipDeviceSynchronize();
  }


  std::vector<gpu_queue_entry> test;
  test.resize(queue_sizes[0]);
  hipMemcpy(test.data(),
            pointers.queue_device_,
            queue_sizes[0] * sizeof(gpu_queue_entry),
            hipMemcpyDeviceToHost);

  gpu_search_results results;
  std::vector<std::size_t> queue_index = {0, 3750000, 7500000, 11250000,
                                          15000000, 18750000, 22500000, 26250000};

  results.gpu_final_queues_.resize(max_transfers + 1);
  for (auto transfers = 0U; transfers <= max_transfers; ++ transfers) {
    results.gpu_final_queues_[transfers].resize(queue_sizes[transfers]);
    CUDA_COPY(results.gpu_final_queues_[transfers].data(),
              pointers.queue_device_ + queue_index[transfers],
              queue_sizes[transfers] * sizeof(gpu_queue_entry),
              hipMemcpyDeviceToHost)
  }

  unsigned result_set_size;
  CUDA_COPY(&result_set_size,
            pointers.result_set_size_device_,
            sizeof(unsigned),
            hipMemcpyDeviceToHost)
  std::cout << result_set_size << std::endl;

  unsigned thread_num = result_set_size;
  unsigned block_num = 1;
  if(thread_num > 32) {
    block_num = (thread_num + 31) / 32;
    thread_num = 32;
  }
  check_dominated<<<block_num, thread_num>>>(pointers);
  hipDeviceSynchronize();

  results.gpu_result_journeys_.resize(result_set_size);
  CUDA_COPY(results.gpu_result_journeys_.data(),
            pointers.result_set_device_,
            result_set_size * sizeof(gpu_tb_journey),
            hipMemcpyDeviceToHost)
  std::cout << results.gpu_result_journeys_.size() << std::endl;

  results.gpu_is_dominated_.resize(result_set_size);
  CUDA_COPY(results.gpu_is_dominated_.data(),
            pointers.is_dominated_device_,
            result_set_size * sizeof(uint8_t),
            hipMemcpyDeviceToHost)
  std::cout << results.gpu_is_dominated_.size() << std::endl;

  return results;
}

gpu_device_pointers allocate_and_copy_on_device(
    std::vector<std::vector<gpu_dest_arrival>> dest_arrs,
    gpu_fws_multimap_arrival_times arrival_times,
    uint16_t total_earliest_arrival,
    uint16_t* line_stop_count,
    std::size_t line_stop_count_size,
    gpu_nested_fws_multimap_transfers transfers,
    uint32_t* trip_to_line,
    std::size_t trip_to_line_size,
    uint16_t start_time,
    uint16_t* first_reachable_stop,
    std::size_t first_reachable_stop_size,
    uint64_t trip_count,
    gpu_queue_entry* initial_queue,
    std::size_t initial_queue_size,
    unsigned max_transfers) {

  gpu_device_pointers pointers;

  std::size_t dest_arrivals_size = dest_arrs.size();

  std::vector<std::size_t> dest_arrivals_index;
  //dest_arrivals_index.resize(dest_arrivals_size + 1);
  dest_arrivals_index.emplace_back(0);

  std::size_t dest_arrs_size = 0;
  for (auto dest_arr : dest_arrs) {
    dest_arrs_size += dest_arr.size();
    dest_arrivals_index.emplace_back(dest_arrs_size);
  }
  CUDA_ALLOC(pointers.dest_arrivals_device_,
             dest_arrs_size * sizeof(gpu_dest_arrival))
  for (auto i = 0; i < dest_arrivals_size; ++i) {
    std::vector<gpu_dest_arrival> dest_arr = dest_arrs[i];
    CUDA_COPY(&pointers.dest_arrivals_device_[dest_arrivals_index[i]],
              dest_arr.data(),
              dest_arr.size() * sizeof(gpu_dest_arrival),
              hipMemcpyHostToDevice)
  }
  CUDA_ALLOC_COPY(pointers.dest_arrivals_index_device_,
                  dest_arrivals_index.data(),
                  dest_arrivals_index.size() * sizeof(std::size_t))

  std::vector<std::size_t> test2;
  test2.resize(dest_arrivals_index.size());
  CUDA_COPY(test2.data(), pointers.dest_arrivals_index_device_,
            dest_arrivals_index.size() * sizeof(std::size_t), hipMemcpyDeviceToHost)
  for(auto i = 0; i < test2.size() - 1; ++i) {
    if (test2[i + 1] > test2[i]) {
      //std::printf("test");
      std::cout << i << std::endl;
    }
  }

  CUDA_ALLOC_COPY(pointers.arrival_times_device_.data_,
            arrival_times.data_,
            arrival_times.index_[*arrival_times.index_size_ - 1] * sizeof(uint16_t))
  CUDA_ALLOC_COPY(pointers.arrival_times_device_.index_,
            arrival_times.index_,
            *arrival_times.index_size_ * sizeof(std::size_t))
  CUDA_ALLOC_COPY(pointers.arrival_times_device_.index_size_,
            &arrival_times.index_size_,
            sizeof(std::size_t))

  CUDA_ALLOC_COPY(pointers.total_earliest_arrival_device_,
                  &total_earliest_arrival,
                  sizeof(uint16_t))

  CUDA_ALLOC_COPY(pointers.line_stop_count_device_,
                  line_stop_count,
                  line_stop_count_size * sizeof(uint16_t))

  CUDA_ALLOC_COPY(pointers.transfers_device.data_,
            transfers.data_,
            transfers.index_[*transfers.index_size_ - 1] * sizeof(gpu_tb_transfer))
  CUDA_ALLOC_COPY(pointers.transfers_device.base_index_,
            transfers.base_index_,
            *transfers.base_index_size_ * sizeof(std::size_t))
  CUDA_ALLOC_COPY(pointers.transfers_device.index_,
            transfers.index_,
            *transfers.index_size_ * sizeof(std::size_t))
  CUDA_ALLOC_COPY(pointers.transfers_device.base_index_size_,
            &transfers.base_index_size_,
            sizeof(std::size_t))
  CUDA_ALLOC_COPY(pointers.transfers_device.index_size_,
            &transfers.index_size_,
            sizeof(std::size_t))

  CUDA_ALLOC_COPY(pointers.trip_to_line_device_,
                  trip_to_line,
                  trip_to_line_size * sizeof(uint32_t))

  CUDA_ALLOC_COPY(pointers.start_time_device_,
                  &start_time,
                  sizeof(uint16_t))

  CUDA_ALLOC_COPY(pointers.first_reachable_stop_device_,
                  first_reachable_stop,
                  first_reachable_stop_size * sizeof(uint16_t))

  CUDA_ALLOC_COPY(pointers.trip_count_device_,
                  &trip_count,
                  sizeof(uint64_t))

  std::size_t queue_size = 30000000 * sizeof(gpu_queue_entry);
  CUDA_ALLOC(pointers.queue_device_,
             queue_size)
  CUDA_COPY(pointers.queue_device_,
            initial_queue,
            initial_queue_size * sizeof(gpu_queue_entry),
            hipMemcpyHostToDevice)

  std::vector<gpu_queue_entry> test;
  test.resize(initial_queue_size);
  hipMemcpy(test.data(), pointers.queue_device_,
             initial_queue_size * sizeof(gpu_queue_entry), hipMemcpyDeviceToHost);

  std::vector<std::size_t> queue_index = {0, 3750000, 7500000, 11250000,
                                          15000000, 18750000, 22500000, 26250000};
  CUDA_ALLOC_COPY(pointers.queue_index_device_,
                  queue_index.data(),
                  queue_index.size() * sizeof(std::size_t))

  std::vector<unsigned> used_queue_sizes(max_transfers + 1);
  used_queue_sizes[0] = initial_queue_size;
  CUDA_ALLOC(pointers.queue_sizes_device_,
             used_queue_sizes.size() * sizeof(unsigned));
  CUDA_COPY(pointers.queue_sizes_device_,
            used_queue_sizes.data(),
            used_queue_sizes.size() * sizeof(unsigned),
            hipMemcpyHostToDevice);

  // TODO(sarah): size enough?
  std::size_t result_set_alloc_num = 1000;
  CUDA_ALLOC(pointers.result_set_device_,
             result_set_alloc_num * sizeof(gpu_tb_journey))
  unsigned zero = 0;
  CUDA_ALLOC_COPY(pointers.result_set_size_device_,
                  &zero,
                  sizeof(unsigned))

  // TODO(sarah): same size as result set above (-> same question)
  std::vector<uint8_t> is_dominated_init(result_set_alloc_num);
  CUDA_ALLOC_COPY(pointers.is_dominated_device_,
                  is_dominated_init.data(),
                  result_set_alloc_num * sizeof(uint8_t))

  return pointers;
}

void free_on_device(gpu_device_pointers pointers) {
  hipFree(pointers.dest_arrivals_device_);
  hipFree(pointers.dest_arrivals_index_device_);
  hipFree(pointers.arrival_times_device_.data_);
  hipFree(pointers.arrival_times_device_.index_);
  hipFree(pointers.arrival_times_device_.index_size_);
  hipFree(pointers.total_earliest_arrival_device_);
  hipFree(pointers.line_stop_count_device_);
  hipFree(pointers.transfers_device.data_);
  hipFree(pointers.transfers_device.base_index_);
  hipFree(pointers.transfers_device.base_index_size_);
  hipFree(pointers.transfers_device.index_);
  hipFree(pointers.transfers_device.index_size_);
  hipFree(pointers.trip_to_line_device_);
  hipFree(pointers.start_time_device_);
  hipFree(pointers.first_reachable_stop_device_);
  hipFree(pointers.trip_count_device_);
  hipFree(pointers.queue_device_);
  hipFree(pointers.queue_index_device_);
  hipFree(pointers.queue_sizes_device_);
  hipFree(pointers.result_set_device_);
  hipFree(pointers.result_set_size_device_);
  hipFree(pointers.is_dominated_device_);
}

} // namespace motis::tripbased